#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hipblas.h>

// This file measures the FLOPS that can be achieved by using gemm functions.
// This is intended to measure performance when the GPU is at different frequencies.

#define RUNTIME 10 // In seconds

template <typename T>
void fill_random(T* arr, int dim_size) {
    for(int i = 0; i < dim_size * dim_size; i++) {
        arr[i] = (T)rand()/RAND_MAX;
    }
}

template <>
void fill_random(__half* arr, int dim_size) {
    for(int i = 0; i < dim_size * dim_size; i++) {
        arr[i] = __float2half(rand()/RAND_MAX);
    }
}



template <typename T>
int gemm(hipblasHandle_t handle, int dim, T d_A, T d_B, T d_C) {
    printf("Unsupported Type\n");
    return -1;
}

template <>
int gemm(hipblasHandle_t handle, int dim, __half *d_A, __half *d_B, __half *d_C) {
    __half alpha = __float2half(1.0f);
    __half beta = __float2half(0.0f);
    int i = 0;
    std::chrono::system_clock::time_point end = std::chrono::system_clock::now() + std::chrono::seconds(RUNTIME);
    while (std::chrono::system_clock::now() < end) {
        i++;
        hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &alpha, d_A, dim, d_B, dim, &beta, d_C, dim);
        hipDeviceSynchronize();
    }
    return i;
}

template <>
int gemm(hipblasHandle_t handle, int dim, float *d_A, float *d_B, float *d_C) {
    float alpha = 1;
    float beta = 0;
    int i = 0;
    std::chrono::system_clock::time_point end = std::chrono::system_clock::now() + std::chrono::seconds(RUNTIME);
    while (std::chrono::system_clock::now() < end) {
        i++;
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &alpha, d_A, dim, d_B, dim, &beta, d_C, dim);
        hipDeviceSynchronize();
    }
    return i;
}

template <>
int gemm(hipblasHandle_t handle, int dim, double *d_A, double *d_B, double *d_C) {
    double alpha = 1;
    double beta = 0;
    int i = 0;
    std::chrono::system_clock::time_point end = std::chrono::system_clock::now() + std::chrono::seconds(RUNTIME);
    while (std::chrono::system_clock::now() < end) {
        i++;
        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &alpha, d_A, dim, d_B, dim, &beta, d_C, dim);
        hipDeviceSynchronize();
    }
    return i;
}



template <typename T>
void benchmark(int min_dim, int max_dim) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    printf("Allocating array... ");
    T* h_A = (T*)malloc(max_dim * max_dim * sizeof(T));
    T* h_B = (T*)malloc(max_dim * max_dim * sizeof(T));
    T* h_C = (T*)malloc(max_dim * max_dim * sizeof(T));
    printf("Done\n");

    printf("Filling with random... ");
    fill_random(h_A, max_dim);
    fill_random(h_B, max_dim);
    printf("Done\n");

    printf("Allocating on GPU... ");
    T *d_A, *d_B, *d_C;
    hipMalloc(&d_A, max_dim * max_dim * sizeof(T));
    hipMalloc(&d_B, max_dim * max_dim * sizeof(T));
    hipMalloc(&d_C, max_dim * max_dim * sizeof(T));
    printf("Done\n");

    printf("Copying to GPU... ");
    hipMemcpy(d_A, h_A, max_dim * max_dim * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, max_dim * max_dim * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, max_dim * max_dim * sizeof(T), hipMemcpyHostToDevice);
    printf("Done\n");


    printf("Running GEMM... \n");
    hipEvent_t gpu_start, gpu_end;
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_end);
    float time_ms;
    double num_flop, final_time, final_flops;
    int num_iterations;
    for (int dim = min_dim; dim <= max_dim; dim *= 2) {
        printf("Matrix %d - ", dim);
        hipEventRecord(gpu_start);

        num_iterations = gemm(handle, dim, d_A, d_B, d_C);

        hipEventRecord(gpu_end);
        hipEventSynchronize(gpu_end);
        hipEventElapsedTime(&time_ms, gpu_start, gpu_end);

        // num_flop is the # of Floating Point Operations that should take place in a SINGLE matrix multiply
        num_flop = (unsigned long long)(dim * dim) * ((unsigned long long)(2 * dim) - 1);
        // final_time is the average time that it takes to do one matrix multiply
        final_time = ((time_ms / 1000.0) / num_iterations);
        // final_flops is number of Floating Point Operations Per Second that were achieved
        final_flops = (num_flop / (double) final_time);
        printf("%f FLOPS (%f seconds, %d iterations)\n", final_flops, (time_ms / 1000.0), num_iterations);
    }
    printf("Done\n");

    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
}

int main() {
    setvbuf(stdout, NULL, _IONBF, 0);

    int min_dim = 128;
    int max_dim = 8192;
    printf("Starting HALF\n");
    benchmark<__half>(min_dim, max_dim);
    printf("Done HALF\n\n");

    printf("Starting FLOAT\n");
    benchmark<float>(min_dim, max_dim);
    printf("Done FLOAT\n\n");

    printf("Starting DOUBLE\n");
    benchmark<double>(min_dim, max_dim);
    printf("Done DOUBLE\n\n");

    return 0;
}
