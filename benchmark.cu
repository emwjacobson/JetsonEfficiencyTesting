#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hipblas.h>

// This file measures the FLOPS that can be achieved by using gemm functions.
// This is intended to measure performance when the GPU is at different frequencies.

#define NUM_ITERATIONS 20
#define EASY_COPY

template <typename T>
void fill_random(T* arr, int dim_size) {
    for(int i = 0; i < dim_size * dim_size; i++) {
        arr[i] = (T)rand()/RAND_MAX;
    }
}

template <>
void fill_random(__half* arr, int dim_size) {
    for(int i = 0; i < dim_size * dim_size; i++) {
        arr[i] = __float2half(rand()/RAND_MAX);
    }
}

template <typename T>
void gemm(hipblasHandle_t handle, int dim, T d_A, T d_B, T d_C) {
    printf("Unsupported Type\n");
}

template <>
void gemm(hipblasHandle_t handle, int dim, __half* d_A, __half* d_B, __half* d_C) {
    printf("Half - %d\n", dim);
    __half alpha = __float2half(1.0f);
    __half beta = __float2half(1.0f);
    hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &alpha, d_A, dim, d_B, dim, &beta, d_C, dim);
}

template <>
void gemm(hipblasHandle_t handle, int dim, float* d_A, float* d_B, float* d_C) {
    printf("Float - %d\n", dim);
    float alpha = 1;
    float beta = 1;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &alpha, d_A, dim, d_B, dim, &beta, d_C, dim);
}

template <>
void gemm(hipblasHandle_t handle, int dim, double* d_A, double* d_B, double* d_C) {
    printf("Double - %d\n", dim);
    double alpha = 1;
    double beta = 1;
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &alpha, d_A, dim, d_B, dim, &beta, d_C, dim);
}

template <typename T>
void benchmark(int min_dim, int max_dim) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    T* h_A = (T*)malloc(max_dim * max_dim * sizeof(T));
    T* h_B = (T*)malloc(max_dim * max_dim * sizeof(T));
    T* h_C = (T*)malloc(max_dim * max_dim * sizeof(T));

    fill_random(h_A, max_dim);
    fill_random(h_B, max_dim);

    T *d_A, *d_B, *d_C;
    hipMalloc(&d_A, max_dim * max_dim * sizeof(T));
    hipMalloc(&d_B, max_dim * max_dim * sizeof(T));
    hipMalloc(&d_C, max_dim * max_dim * sizeof(T));

    hipMemcpy(d_A, h_A, max_dim * max_dim * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, max_dim * max_dim * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, max_dim * max_dim * sizeof(T), hipMemcpyHostToDevice);

    for (int dim = min_dim; dim <= max_dim; dim *= 2) {
        gemm<T>(handle, dim, d_A, d_B, d_C);
    }
}

int main() {
    setvbuf(stdout, NULL, _IONBF, 0);

    int min_dim = 128;
    int max_dim = 8192;
    benchmark<__half>(min_dim, max_dim);
    benchmark<float>(min_dim, max_dim);
    benchmark<double>(min_dim, max_dim);

    return 0;
}
