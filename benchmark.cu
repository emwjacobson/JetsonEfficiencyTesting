#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <sys/socket.h>
#include <arpa/inet.h>
#include <thread>
#include <vector>
#include "jetson_clocks.hpp/jetson_clocks.hpp"

// This file measures the FLOPS that can be achieved by using gemm functions.
// This is intended to measure performance when the GPU is at different frequencies.

#define RUNTIME 20 // In seconds
#define USE_SOCKET

template <typename T>
void fill_random(T* arr, int dim_size) {
    for(int i = 0; i < dim_size * dim_size; i++) {
        arr[i] = (T)rand()/RAND_MAX;
    }
}

template <>
void fill_random(__half* arr, int dim_size) {
    for(int i = 0; i < dim_size * dim_size; i++) {
        arr[i] = __float2half(rand()/RAND_MAX);
    }
}


// START TENSOR FUNCTIONS

template <typename T>
int gemm(hipblasHandle_t handle, int dim, T d_A, T d_B, T d_C) {
    printf("Unsupported Type\n");
    return -1;
}

template <>
int gemm(hipblasHandle_t handle, int dim, __half *d_A, __half *d_B, __half *d_C) {
    __half alpha = __float2half(1.0f);
    __half beta = __float2half(0.0f);
    int i = 0;
    std::chrono::system_clock::time_point end = std::chrono::system_clock::now() + std::chrono::seconds(RUNTIME);
    while (std::chrono::system_clock::now() < end) {
        i++;
        hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &alpha, d_A, dim, d_B, dim, &beta, d_C, dim);
        hipDeviceSynchronize();
    }
    return i;
}

template <>
int gemm(hipblasHandle_t handle, int dim, float *d_A, float *d_B, float *d_C) {
    float alpha = 1;
    float beta = 0;
    int i = 0;
    std::chrono::system_clock::time_point end = std::chrono::system_clock::now() + std::chrono::seconds(RUNTIME);
    while (std::chrono::system_clock::now() < end) {
        i++;
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &alpha, d_A, dim, d_B, dim, &beta, d_C, dim);
        hipDeviceSynchronize();
    }
    return i;
}

template <>
int gemm(hipblasHandle_t handle, int dim, double *d_A, double *d_B, double *d_C) {
    double alpha = 1;
    double beta = 0;
    int i = 0;
    std::chrono::system_clock::time_point end = std::chrono::system_clock::now() + std::chrono::seconds(RUNTIME);
    while (std::chrono::system_clock::now() < end) {
        i++;
        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &alpha, d_A, dim, d_B, dim, &beta, d_C, dim);
        hipDeviceSynchronize();
    }
    return i;
}

// END NON-TENSOR FUNCTIONS


// START TENSOR FUNCTIONS

template <typename T>
int gemm_tensor(hipblasHandle_t handle, int dim, T d_A, T d_B, T d_C) {
    printf("Unsupported Type\n");
    return -1;
}

template <>
int gemm_tensor(hipblasHandle_t handle, int dim, __half *d_A, __half *d_B, __half *d_C) {
    __half alpha = __float2half(1.0f);
    __half beta = __float2half(0.0f);
    int i = 0;
    std::chrono::system_clock::time_point end = std::chrono::system_clock::now() + std::chrono::seconds(RUNTIME);
    while (std::chrono::system_clock::now() < end) {
        i++;
        hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &alpha, d_A, HIP_R_16F, dim, d_B, HIP_R_16F, dim, &beta, d_C, HIP_R_16F, dim, HIP_R_16F, HIPBLAS_GEMM_DEFAULT);
        hipDeviceSynchronize();
    }
    return i;
}

template <>
int gemm_tensor(hipblasHandle_t handle, int dim, float *d_A, float *d_B, float *d_C) {
    float alpha = 1;
    float beta = 0;
    int i = 0;
    std::chrono::system_clock::time_point end = std::chrono::system_clock::now() + std::chrono::seconds(RUNTIME);
    while (std::chrono::system_clock::now() < end) {
        i++;
        hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &alpha, d_A, HIP_R_32F, dim, d_B, HIP_R_32F, dim, &beta, d_C, HIP_R_32F, dim, HIP_R_32F, HIPBLAS_GEMM_DEFAULT);
        hipDeviceSynchronize();
    }
    return i;
}

template <>
int gemm_tensor(hipblasHandle_t handle, int dim, double *d_A, double *d_B, double *d_C) {
    double alpha = 1;
    double beta = 0;
    int i = 0;
    std::chrono::system_clock::time_point end = std::chrono::system_clock::now() + std::chrono::seconds(RUNTIME);
    while (std::chrono::system_clock::now() < end) {
        i++;
        hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &alpha, d_A, HIP_R_64F, dim, d_B, HIP_R_64F, dim, &beta, d_C, HIP_R_64F, dim, HIP_R_64F, HIPBLAS_GEMM_DEFAULT);
        hipDeviceSynchronize();
    }
    return i;
}

// END TENSOR FUNCTIONS


template <typename T>
std::string get_datatype(T* type) {
    return std::string("Unknown Datatype");
}

template <>
std::string get_datatype(__half* type) {
    return std::string("half");
}
template <>
std::string get_datatype(float* type) {
    return std::string("float");
}
template <>
std::string get_datatype(double* type) {
    return std::string("double");
}


template <typename T>
void benchmark(int sock, int min_dim, int max_dim) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    printf("Allocating array... ");
    T* h_A = (T*)malloc(max_dim * max_dim * sizeof(T));
    T* h_B = (T*)malloc(max_dim * max_dim * sizeof(T));
    T* h_C = (T*)malloc(max_dim * max_dim * sizeof(T));
    printf("Done\n");

    printf("Filling with random... ");
    fill_random(h_A, max_dim);
    fill_random(h_B, max_dim);
    printf("Done\n");

    printf("Allocating on GPU... ");
    T *d_A, *d_B, *d_C;
    hipMalloc(&d_A, max_dim * max_dim * sizeof(T));
    hipMalloc(&d_B, max_dim * max_dim * sizeof(T));
    hipMalloc(&d_C, max_dim * max_dim * sizeof(T));
    printf("Done\n");

    printf("Copying to GPU... ");
    hipMemcpy(d_A, h_A, max_dim * max_dim * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, max_dim * max_dim * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, max_dim * max_dim * sizeof(T), hipMemcpyHostToDevice);
    printf("Done\n");


    printf("Running GEMM... \n");
    hipEvent_t gpu_start, gpu_end;
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_end);
    float time_ms;
    double num_flop, final_time, final_flops;
    int num_iterations;
    std::string msg;

    // Test tensor core and non-tensor core
    // https://forums.developer.nvidia.com/t/how-to-confirm-whether-tensor-core-is-working-or-not/70263/8

    // This is the "non-tensor" version using the individual cublas<t>gemm functions
    for (int dim = min_dim; dim <= max_dim; dim += 128) {
        printf("Matrix %d (Non-tensor) - ", dim);
        hipEventRecord(gpu_start);

        #ifdef USE_SOCKET
        // START,datatype,dim_size,nontensor
        // eg. START,half,256,nontensor
        msg = "START," + get_datatype(h_A) + "," + std::to_string(dim) + ",nontensor," + std::to_string(jetson_clocks::get_gpu_cur_freq());
        send(sock, msg.c_str(), strlen(msg.c_str()), 0);
        #endif

        num_iterations = gemm(handle, dim, d_A, d_B, d_C);

        hipEventRecord(gpu_end);
        hipEventSynchronize(gpu_end);
        hipEventElapsedTime(&time_ms, gpu_start, gpu_end);

        // num_flop is the # of Floating Point Operations that should take place in a SINGLE matrix multiply
        num_flop = (unsigned long long)(dim * dim) * ((unsigned long long)(2 * dim) - 1);
        // final_time is the average time that it takes to do one matrix multiply
        final_time = ((time_ms / 1000.0) / num_iterations);
        // final_flops is number of Floating Point Operations Per Second that were achieved
        final_flops = (num_flop / (double) final_time);
        printf("%f FLOPS (%f seconds, %d iterations)\n", final_flops, (time_ms / 1000.0), num_iterations);

        #ifdef USE_SOCKET
        msg = "DONE," + std::to_string(final_flops);
        send(sock, msg.c_str(), strlen(msg.c_str()), 0);
        #endif

        printf("Sleeping between tests...\n");
        std::this_thread::sleep_for(std::chrono::milliseconds(2000));
    }

    // This is the "tensor" version using hipblasGemmEx
    for (int dim = min_dim; dim <= max_dim; dim += 128) {
        printf("Matrix %d (Tensor) - ", dim);
        hipEventRecord(gpu_start);

        #ifdef USE_SOCKET
        // START,datatype,dim_size,tensor
        // eg. START,half,256,tensor
        msg = "START," + get_datatype(h_A) + "," + std::to_string(dim) + ",tensor," + std::to_string(jetson_clocks::get_gpu_cur_freq());
        send(sock, msg.c_str(), strlen(msg.c_str()), 0);
        #endif

        num_iterations = gemm_tensor(handle, dim, d_A, d_B, d_C);

        hipEventRecord(gpu_end);
        hipEventSynchronize(gpu_end);
        hipEventElapsedTime(&time_ms, gpu_start, gpu_end);

        // num_flop is the # of Floating Point Operations that should take place in a SINGLE matrix multiply
        num_flop = (unsigned long long)(dim * dim) * ((unsigned long long)(2 * dim) - 1);
        // final_time is the average time that it takes to do one matrix multiply
        final_time = ((time_ms / 1000.0) / num_iterations);
        // final_flops is number of Floating Point Operations Per Second that were achieved
        final_flops = (num_flop / (double) final_time);
        printf("%f FLOPS (%f seconds, %d iterations)\n", final_flops, (time_ms / 1000.0), num_iterations);

        #ifdef USE_SOCKET
        msg = "DONE," + std::to_string(final_flops);
        send(sock, msg.c_str(), strlen(msg.c_str()), 0);
        #endif

        printf("Sleeping between tests...\n");
        std::this_thread::sleep_for(std::chrono::milliseconds(2000));
    }

    printf("Done\n");

    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
}

int connect_socket() {
    struct sockaddr_in serv_addr;
    char buffer[256];
    char * errorMsg;
    int sock;
    if ((sock = socket(AF_INET, SOCK_STREAM, 0)) < 0) {
        printf("Socket Creation Error\n");
        return -1;
    }
    serv_addr.sin_family = AF_INET;
    serv_addr.sin_port = htons(8888);
    if (inet_pton(AF_INET, "192.168.1.8", &serv_addr.sin_addr) <= 0) {
        printf("Invalid address\n");
        return -1;
    }
    if (connect(sock, (struct sockaddr *)&serv_addr, sizeof(serv_addr)) < 0) {
        errorMsg = strerror_r(errno, buffer, 256);
        printf("Connection Failed: %s\n", errorMsg);

        return -1;
    }
    return sock;
}

void benchmark_datatypes(int sock, int min_dim, int max_dim) {
    printf("Starting HALF\n");
    benchmark<__half>(sock, min_dim, max_dim);
    printf("Done HALF\n\n");

    printf("Long sleep between datatypes...\n");
    std::this_thread::sleep_for(std::chrono::milliseconds(10000));

    printf("Starting FLOAT\n");
    benchmark<float>(sock, min_dim, max_dim);
    printf("Done FLOAT\n\n");

    printf("Long sleep between datatypes...\n");
    std::this_thread::sleep_for(std::chrono::milliseconds(10000));

    printf("Starting DOUBLE\n");
    benchmark<double>(sock, min_dim, max_dim);
    printf("Done DOUBLE\n\n");
}

int main() {
    setvbuf(stdout, NULL, _IONBF, 0);

    int min_dim = 128;
    int max_dim = 2048;

    #ifdef USE_SOCKET
    printf("Connecting to server... ");
    int sock = connect_socket();
    if (sock == -1) {
        printf("Error connecting to socket server.\n");
        return -1;
    }
    printf("Connected\n");
    #endif

    #ifdef USE_SOCKET
    benchmark_datatypes(sock, min_dim, max_dim);
    #else
    benchmark_datatypes(0, min_dim, max_dim);
    #endif

    return 0;
}
